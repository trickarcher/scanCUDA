#include "hip/hip_runtime.h"
#include "awkward_offset_arrays.h"
#include <iostream>
#include "assert.h"
namespace awkward {

template<typename T, typename C>
__global__ void awkward_listarray_compact_offsets_cuda(T *d_in,
                                                       T *d_out,
                                                       T *d_tooffsets,
                                                       int64_t curr_step,
                                                       int64_t total_steps,
                                                       int64_t stride,
                                                       int64_t length,
                                                       bool in_out_flag) {

  int block_id = blockIdx.x + blockIdx.y * gridDim.x + gridDim.x * gridDim.y * blockIdx.z;

  int thread_id = block_id * blockDim.x + threadIdx.x;
  T sum = 0;

  if (thread_id < length) {
    if (!in_out_flag) {
      if (thread_id < stride) {
        sum = d_out[thread_id];
        d_in[thread_id] = sum;
      } else {
        sum = d_out[thread_id] + d_out[thread_id - stride];
        d_in[thread_id] = sum;
      }
    } else {
      if (thread_id < stride) {
        sum = d_in[thread_id];
        d_out[thread_id] = sum;
      } else {
        sum = d_in[thread_id] + d_in[thread_id - stride];
        d_out[thread_id] = sum;
      }
    }

    if (curr_step == total_steps) {
      d_tooffsets[thread_id] = sum;
    }
  }
}

template<typename T, typename C>
__global__ void awkward_listarray_compute_offsets_cuda(T *d_tooffsets,
                                                       C *d_fromstarts,
                                                       C *d_fromstops,
                                                       int64_t startoffset,
                                                       int64_t stopoffset,
                                                       int64_t length) {

  auto block_id = blockIdx.x + blockIdx.y * gridDim.x + gridDim.x * gridDim.y * blockIdx.z;
  auto thread_id = block_id * blockDim.x + threadIdx.x;

  if (thread_id < length) {
    if (d_fromstops[thread_id + stopoffset] > d_fromstarts[thread_id + startoffset])
      d_tooffsets[thread_id + 1] = d_fromstops[thread_id + stopoffset] - d_fromstarts[thread_id + startoffset];
    else {
      assert("Invalid!");
//      ~AwkwardOffsetArrayCuda();
    }
  }
}

template<typename T, typename C>
void AwkwardOffsetArrayCuda<T, C>::compute_offsets_kernel_wrapper() {
  awkward_listarray_compute_offsets_cuda<T, C><<<blocks_per_grid, threads_per_block>>>(d_tooffsets,
                                                                                       d_fromstarts,
                                                                                       d_fromstops,
                                                                                       startoffset,
                                                                                       stopoffset,
                                                                                       length);
  HANDLE_ERROR(hipFree(d_fromstarts));
  HANDLE_ERROR(hipFree(d_fromstops));
}

template<typename T, typename C>
void AwkwardOffsetArrayCuda<T, C>::compact_offsets_kernel_wrapper() {
  int64_t stride = 1;
  int total_steps = ceil(log2(static_cast<float>(length)));
  for (size_t curr_step = 1; curr_step <= total_steps; curr_step++) {
    bool in_out_flag = (curr_step % 2) != 0;
    awkward_listarray_compact_offsets_cuda<T, C><<<blocks_per_grid, threads_per_block>>>(d_in,
                                                                                         d_out,
                                                                                         d_tooffsets,
                                                                                         curr_step,
                                                                                         total_steps,
                                                                                         stride,
                                                                                         length,
                                                                                         in_out_flag);
    stride = stride * 2;
  }

  HANDLE_ERROR(hipMemcpy(tooffsets, d_tooffsets, SIZE_T, hipMemcpyDeviceToHost));
  this->tooffsets[length] += this->tooffsets[length - 1];

  HANDLE_ERROR(hipFree(d_tooffsets));
  HANDLE_ERROR(hipFree(d_in));
  HANDLE_ERROR(hipFree(d_out));
}
template
class AwkwardOffsetArrayCuda<int64_t, int32_t>;

template
class AwkwardOffsetArrayCuda<int16_t, int8_t>;

template
class AwkwardOffsetArrayCuda<int32_t, int16_t>;

template
class AwkwardOffsetArrayCuda<int, int>;

template
class AwkwardOffsetArrayCuda<int64_t, int8_t>;
}